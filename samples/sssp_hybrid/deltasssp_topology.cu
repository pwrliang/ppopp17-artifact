#include "hip/hip_runtime.h"
// Groute: An Asynchronous Multi-GPU Programming Framework
// http://www.github.com/groute/groute
// Copyright (c) 2017, A. Barak
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice,
//   this list of conditions and the following disclaimer.
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
// * Neither the names of the copyright holders nor the names of its
//   contributors may be used to endorse or promote products derived from this
//   software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.

#include <vector>
#include <algorithm>
#include <thread>
#include <memory>
#include <random>
#include <>
#include <gflags/gflags.h>

#include <groute/device/cta_scheduler.cuh>
#include <groute/graphs/csr_graph.h>
#include <groute/dwl/distributed_worklist.cuh>
#include <groute/dwl/workers.cuh>
#include <utils/cuda_utils.h>
#include <utils/graphs/traversal.h>
#include <utils/balancer.h>
#include "sssp_common.h"

#define GTID (blockIdx.x * blockDim.x + threadIdx.x)
#define FILTER_THRESHOLD 0.0000000001
DECLARE_double(wl_alloc_factor_local);
DECLARE_int32(source_node);
DECLARE_int32(grid_size);
DECLARE_int32(block_size);
DECLARE_int32(mode);
DECLARE_int32(source_node);
DECLARE_int32(async_to_sync);
DECLARE_int32(sync_to_async);
DECLARE_bool(force_sync);
DECLARE_bool(force_async);


const distance_t INF = UINT_MAX;
namespace sssp_expr {
    const distance_t IDENTITY_ELEMENT = UINT_MAX;

    struct Algo {
        static const char *Name() { return "SSSP"; }
    };


    __inline__ __device__ uint32_t warpReduce(uint32_t localSum) {
        localSum += __shfl_xor_sync(0xfffffff, localSum, 16);
        localSum += __shfl_xor_sync(0xfffffff, localSum, 8);
        localSum += __shfl_xor_sync(0xfffffff, localSum, 4);
        localSum += __shfl_xor_sync(0xfffffff, localSum, 2);
        localSum += __shfl_xor_sync(0xfffffff, localSum, 1);

        return localSum;
    }

    template<template<typename> class TDistanceDatum>
    __device__ void SSSPCheck__Single__(TDistanceDatum<distance_t> current_ranks,
                                        distance_t *block_sum_buffer, distance_t *rtn_sum) {
        unsigned tid = TID_1D;
        unsigned nthreads = TOTAL_THREADS_1D;
        int laneIdx = threadIdx.x % warpSize;
        int warpIdx = threadIdx.x / warpSize;
        const int SMEMDIM = blockDim.x / warpSize;
        __shared__ distance_t smem[32];

        uint32_t work_size = current_ranks.size;
        distance_t local_sum = 0;

        for (uint32_t node = 0 + tid; node < work_size; node += nthreads) {
            distance_t dist = current_ranks[node];
            if (dist != IDENTITY_ELEMENT)
                local_sum += dist;
        }

        local_sum = warpReduce(local_sum);

        if (laneIdx == 0)
            smem[warpIdx] = local_sum;
        __syncthreads();

        local_sum = (threadIdx.x < SMEMDIM) ? smem[threadIdx.x] : 0;

        if (warpIdx == 0)
            local_sum = warpReduce(local_sum);

        if (threadIdx.x == 0) {
            block_sum_buffer[blockIdx.x] = local_sum;
        }

        if (tid == 0) {
            uint32_t sum = 0;
            for (int bid = 0; bid < gridDim.x; bid++) {

                sum += block_sum_buffer[bid];
            }
            *rtn_sum = sum;
        }
    }

//    template<
//            template<typename> class WorkList,
//            typename TGraph, typename TWeightDatum,
//            template<typename> class TDistanceDatum,
//            template<typename> class TDistanceDeltaDatum>
//    __device__ void SSSPAsync(
//            const WorkList<index_t> &work_source,
//            WorkList<index_t> &work_immediate_target,
//            WorkList<index_t> &work_later_target,
//            const distance_t priority_threshold,
//            const TGraph &graph,
//            const TWeightDatum &edge_weights,
//            TDistanceDatum<distance_t> &node_distances,
//            TDistanceDeltaDatum<distance_t> &node_distances_delta) {
//        uint32_t tid = TID_1D;
//        uint32_t nthreads = TOTAL_THREADS_1D;
//
//
//        uint32_t work_size = work_source.count();
//
//        for (uint32_t i = 0 + tid; i < work_size; i += nthreads) {
//
//            index_t node = work_source.read(i);
//
//            distance_t old_value = node_distances[node];
//            distance_t old_delta = atomicExch(node_distances_delta.get_item_ptr(node), IDENTITY_ELEMENT);
//            distance_t new_value = min(old_value, old_delta);
//
//            if (new_value != old_value) {
//                for (index_t edge = graph.begin_edge(node), end_edge = graph.end_edge(node); edge < end_edge; ++edge) {
//                    index_t dest = graph.edge_dest(edge);
//                    distance_t weight = edge_weights.get_item(edge);
//                    distance_t new_delta = old_delta + weight;
//                    distance_t before = atomicMin(node_distances_delta.get_item_ptr(dest), new_delta);
//
//                    if (new_delta < before) {
//                        if (new_delta < priority_threshold)
//                            work_immediate_target.append_warp(dest);
//                        else
//                            work_later_target.append_warp(dest);
//                    }
//                }
//            }
//        }
//    }


    template<
            typename WorkSource,
            typename TGraph, typename TWeightDatum,
            template<typename> class TDistanceDatum,
            template<typename> class TDistanceDeltaDatum>
    __global__
    void SSSPAsyncCTA(
            uint32_t *send_count,
            int *active_count,
            const WorkSource work_source,
            const TGraph graph,
            const TWeightDatum edge_weights,
            TDistanceDatum<distance_t> node_distances,
            TDistanceDeltaDatum<distance_t> node_distances_delta) {
        uint32_t tid = TID_1D;
        uint32_t nthreads = TOTAL_THREADS_1D;
        uint32_t work_size = work_source.get_size();
        uint32_t work_size_rup =
                round_up(work_size, blockDim.x) * blockDim.x; // we want all threads in active blocks to enter the loop
        bool updated = false;


        for (uint32_t i = 0 + tid; i < work_size_rup; i += nthreads) {
            groute::dev::np_local<distance_t> np_local = {0, 0, 0};

            if (i < work_size) {
                index_t node = work_source.get_work(i);

                distance_t old_value = node_distances[node];
                distance_t old_delta = atomicExch(node_distances_delta.get_item_ptr(node), IDENTITY_ELEMENT);
                distance_t new_value = min(old_value, old_delta);

                if (new_value < old_value) {
                    node_distances[node] = new_value;

                    np_local.start = graph.begin_edge(node);
                    np_local.size = graph.end_edge(node) - np_local.start;
                    np_local.meta_data = old_delta;
                    updated = true;
                }
            }

            groute::dev::CTAWorkScheduler<distance_t>::template schedule(
                    np_local,
                    [&send_count, &graph, &edge_weights, &node_distances_delta](
                            index_t edge,
                            index_t size,
                            distance_t old_delta) {
                        index_t dest = graph.edge_dest(edge);
                        distance_t weight = edge_weights.get_item(edge);
                        distance_t new_delta = old_delta + weight;
                        distance_t before_update = atomicMin(node_distances_delta.get_item_ptr(dest), new_delta);
                        atomicAdd(send_count, 1);
                    });
        }
        if (updated)
            atomicAdd(active_count, 1);
    }

    //for later nodes, even though...delta > value, but as long as delta != INF, we stil have to send delta to the neighbors.
    template<
            typename WorkSource,
            typename TGraph, typename TWeightDatum,
            template<typename> class TDistanceDatum,
            template<typename> class TDistanceDeltaDatum>
    __global__
    void SSSPSyncCTA(
            uint32_t *send_count,
            int *active_count,
            const WorkSource work_source,
            index_t iteration,
            const TGraph graph,
            TWeightDatum edge_weights,
            TDistanceDatum<distance_t> node_distances,
            TDistanceDeltaDatum<distance_t> node_distances_delta,
            TDistanceDeltaDatum<distance_t> node_distances_last_delta) {
        uint32_t tid = TID_1D;
        uint32_t nthreads = TOTAL_THREADS_1D;
        uint32_t work_size = work_source.get_size();
        uint32_t work_size_rup =
                round_up(work_size, blockDim.x) * blockDim.x; // we want all threads in active blocks to enter the loop
        bool updated = false;

        for (uint32_t i = 0 + tid; i < work_size_rup; i += nthreads) {
            groute::dev::np_local<distance_t> np_local = {0, 0, 0};

            if (i < work_size) {
                index_t node = work_source.get_work(i);
                distance_t old_value = node_distances[node];
                distance_t old_delta;

                if (iteration % 2 == 0) {
                    old_delta = atomicExch(node_distances_delta.get_item_ptr(node), IDENTITY_ELEMENT);
                } else {
                    old_delta = atomicExch(node_distances_last_delta.get_item_ptr(node), IDENTITY_ELEMENT);
                }

                distance_t new_value = min(old_value, old_delta);

                if (new_value < old_value) {
                    node_distances[node] = new_value;

                    np_local.start = graph.begin_edge(node);
                    np_local.size = graph.end_edge(node) - np_local.start;
                    np_local.meta_data = old_delta;
                    updated = true;
                }
            }

            groute::dev::CTAWorkScheduler<distance_t>::template schedule(
                    np_local,
                    [&send_count, &iteration, &graph, &edge_weights, &node_distances_delta, &node_distances_last_delta](
                            index_t edge,
                            index_t size,
                            distance_t old_delta) {
                        index_t dest = graph.edge_dest(edge);
                        distance_t weight = edge_weights.get_item(edge);
                        distance_t new_delta = old_delta + weight;
                        distance_t before_update;

                        if (iteration % 2 == 0) {
                            before_update = atomicMin(node_distances_last_delta.get_item_ptr(dest), new_delta);
                        } else {
                            before_update = atomicMin(node_distances_delta.get_item_ptr(dest), new_delta);
                        }
                        atomicAdd(send_count, 1);
                    });
        }
        if (updated)
            atomicAdd(active_count, 1);
    }

    template<typename T>
    __device__ void swap(T &a, T &b) {
        T tmp = a;
        a = b;
        b = tmp;
    }

    //try to use topologoy

    template<typename WorkSource,
            typename TGraph,
            template<typename> class TWeightDatum,
            template<typename> class TDistanceDatum,
            template<typename> class TDistanceDeltaDatum>
    __global__ void SSSPControlHybrid__Single__(uint32_t async_to_sync,
                                                uint32_t sync_to_async,
                                                int *running_flag,
                                                hipcub::GridBarrier grid_barrier,
                                                WorkSource work_source,
                                                const TGraph graph,
                                                const TWeightDatum<distance_t> edge_weights,
                                                TDistanceDatum<distance_t> node_distances,
                                                TDistanceDeltaDatum<distance_t> node_distances_delta,
                                                TDistanceDeltaDatum<distance_t> node_distances_last_delta) {

        uint32_t tid = TID_1D;
        uint32_t nthreads = TOTAL_THREADS_1D;
        TDistanceDeltaDatum<distance_t> *available_delta = &node_distances_delta;


        //Async->Sync->Async
        //Async -> Sync, no limitation
        //Sync -> Async, iteration % 2 == 1


        if (tid == 0) {
            printf("CALL SSSPControl%s__Single__ InitPrio:\n", "Hybrid");
        }
//
        uint32_t iteration = 0;
        bool updated;

        while (*running_flag) {
            if (true || iteration < async_to_sync || iteration >= sync_to_async) {
//                updated = SSSPAsyncCTA(work_source,
//                                       graph,
//                                       edge_weights,
//                                       node_distances,
//                                       *available_delta);
            } else {
                updated = SSSPSyncCTA(work_source,
                                      iteration,
                                      graph,
                                      edge_weights,
                                      node_distances,
                                      node_distances_delta,
                                      node_distances_last_delta);
                if (iteration % 2 == 0) {
                    available_delta = &node_distances_last_delta;
                }
            }

            if (tid == 0) {
                *running_flag = 0;
            }
            grid_barrier.Sync();

            if (updated) {
                int running_threads = atomicAdd(running_flag, 1);
                //printf("running threads:%d\n", running_threads);
            }

            iteration++;
            grid_barrier.Sync();
        }

        if (tid == 0) {
            printf("Total iterations: %d\n", iteration);
        }


        for (uint32_t i = 0 + tid; i < graph.nnodes; i += nthreads) {
            assert(node_distances_delta[i] == IDENTITY_ELEMENT &&
                   node_distances_last_delta[i] == IDENTITY_ELEMENT);
        }
    }


    template<template<typename> class DistanceDatum,
            template<typename> class DistanceDeltaDatum>
    __global__ void
    SSSPInit(index_t source, DistanceDatum<distance_t> distances, DistanceDeltaDatum<distance_t> delta_distances,
             DistanceDeltaDatum<distance_t> last_delta_distances,
             int nnodes) {
        int tid = GTID;
        if (tid < nnodes) {
            distances[tid] = IDENTITY_ELEMENT;
            last_delta_distances[tid] = IDENTITY_ELEMENT;
            delta_distances[tid] = (tid == source ? 0 : IDENTITY_ELEMENT);
        }
    }


    template<
            typename TGraph,
            template<typename> class TWeightDatum,
            template<typename> class TDistanceDatum,
            template<typename> class TDistanceDeltaDatum>
    struct Problem {
        TGraph m_graph;
        TWeightDatum<distance_t> m_weights_datum;
        TDistanceDatum<distance_t> m_distances_datum;
        TDistanceDeltaDatum<distance_t> m_distances_delta_datum;
        TDistanceDeltaDatum<distance_t> m_distances_last_delta_datum;
        distance_t m_priority_threshold;
    public:
        Problem(const TGraph &graph, const TWeightDatum<distance_t> &weights_datum,
                const TDistanceDatum<distance_t> &distances_datum,
                const TDistanceDeltaDatum<distance_t> &distances_delta_datum,
                const TDistanceDeltaDatum<distance_t> &distances_last_delta_datum,
                const distance_t priority_threshold) :
                m_graph(graph), m_weights_datum(weights_datum), m_distances_datum(distances_datum),
                m_distances_delta_datum(distances_delta_datum),
                m_distances_last_delta_datum(distances_last_delta_datum),
                m_priority_threshold(priority_threshold) {
        }

        void Init(groute::Stream &stream) const {
            index_t source_node = min(max(0, FLAGS_source_node), m_graph.nnodes - 1);

            dim3 grid_dims, block_dims;
            KernelSizing(grid_dims, block_dims, m_distances_datum.size);

            Marker::MarkWorkitems(m_distances_datum.size, "SSSPInit");

            SSSPInit << < grid_dims, block_dims, 0, stream.hip_stream >> > (source_node,
                    m_distances_datum, m_distances_delta_datum, m_distances_last_delta_datum, m_distances_datum.size);
        }
    };

}


bool SSSPExpr() {
    typedef sssp_expr::Problem<groute::graphs::dev::CSRGraph, groute::graphs::dev::GraphDatum, groute::graphs::dev::GraphDatum, groute::graphs::dev::GraphDatum> Problem;

    utils::traversal::Context<sssp_expr::Algo> context(1);
    context.configuration.verbose = FLAGS_verbose;
    context.configuration.trace = FLAGS_trace;
    groute::graphs::single::CSRGraphAllocator dev_graph_allocator(context.host_graph);

    context.SetDevice(0);

    groute::graphs::single::EdgeInputDatum<distance_t> edge_weights;
    groute::graphs::single::NodeOutputDatum<distance_t> node_distances;
    groute::graphs::single::NodeOutputDatum<distance_t> node_delta_distances;
    groute::graphs::single::NodeOutputDatum<distance_t> node_last_delta_distances;

    dev_graph_allocator.AllocateDatumObjects(edge_weights, node_distances, node_delta_distances,
                                             node_last_delta_distances);

    context.SyncDevice(0);

    size_t max_work_size = context.host_graph.nedges * FLAGS_wl_alloc_factor_local;

    groute::Stream stream = context.CreateStream(0);


    Problem problem(dev_graph_allocator.DeviceObject(), edge_weights.DeviceObject(), node_distances.DeviceObject(),
                    node_delta_distances.DeviceObject(), node_last_delta_distances.DeviceObject(), FLAGS_prio_delta);

    problem.Init(stream);
    stream.Sync();


    int occupancy_per_MP = FLAGS_grid_size;
//    hipOccupancyMaxActiveBlocksPerMultiprocessor(&occupancy_per_MP,
//                                                  sssp_expr::SSSPControl__Single__<groute::dev::Queue,
//                                                          groute::graphs::dev::CSRGraph,
//                                                          groute::graphs::dev::GraphDatum,
//                                                          groute::graphs::dev::GraphDatum,
//                                                          groute::graphs::dev::GraphDatum>,
//                                                  FLAGS_block_size, 0);

    hipcub::GridBarrierLifetime grid_barrier;

    grid_barrier.Setup(occupancy_per_MP);

    printf("grid size %d block size %d\n", occupancy_per_MP, FLAGS_block_size);


    utils::SharedArray<distance_t> block_sum_buffer(FLAGS_grid_size);

    auto dev_graph = dev_graph_allocator.DeviceObject();

    utils::SharedValue<int> running_flag;
    utils::SharedValue<uint32_t> send_count;
    running_flag.set_val_H2D(1);
    send_count.set_val_H2D(0);

    Stopwatch sw(true);
    dim3 grid_dim, block_dim;
    int iteration = 0;
    auto *available_delta = &node_delta_distances;

    int mode;

    assert(!(FLAGS_force_sync && FLAGS_force_async));

    while (running_flag.get_val_D2H()) {
        KernelSizing(grid_dim, block_dim, dev_graph.owned_nnodes());
        running_flag.set_val_H2D(0);

        if (FLAGS_force_async)
            goto async;
        else if (FLAGS_force_sync)
            goto sync;

        if (iteration < FLAGS_async_to_sync || iteration > FLAGS_sync_to_async) {
            async:
            sssp_expr::SSSPAsyncCTA << < grid_dim, block_dim, 0, stream.hip_stream >> > (send_count.dev_ptr,
                    running_flag.dev_ptr,
                    groute::dev::WorkSourceRange<index_t>
                            (dev_graph.owned_start_node(), dev_graph.owned_nnodes()),
                    dev_graph,
                    edge_weights.DeviceObject(),
                    node_distances.DeviceObject(),
                    available_delta->DeviceObject());
            mode = 1;
        } else {
            sync:
            sssp_expr::SSSPSyncCTA << < grid_dim, block_dim, 0, stream.hip_stream >> > (send_count.dev_ptr,
                    running_flag.dev_ptr,
                    groute::dev::WorkSourceRange<index_t>
                            (dev_graph.owned_start_node(), dev_graph.owned_nnodes()),
                    iteration,
                    dev_graph,
                    edge_weights.DeviceObject(),
                    node_distances.DeviceObject(),
                    node_delta_distances.DeviceObject(),
                    node_last_delta_distances.DeviceObject());
            if (iteration % 2 == 0)
                available_delta = &node_last_delta_distances;
            mode = 0;
        }

//    sssp_expr::SSSPControlHybrid__Single__
//            << < occupancy_per_MP, FLAGS_block_size, 0, stream.hip_stream >> >
//                                                        (FLAGS_async_to_sync,
//                                                                FLAGS_sync_to_async,
//                                                                running_flag.dev_ptr,
//                                                                grid_barrier,
//                                                                groute::dev::WorkSourceRange<index_t>(
//                                                                        dev_graph.owned_start_node(),
//                                                                        dev_graph.owned_nnodes()),
//                                                                dev_graph,
//                                                                edge_weights.DeviceObject(),
//                                                                node_distances.DeviceObject(),
//                                                                node_delta_distances.DeviceObject(),
//                                                                node_last_delta_distances.DeviceObject());
        stream.Sync();
        iteration++;
//        VLOG(0)
//        << (mode == 1 ? "Async" : "Sync") << "iter: " << iteration << " active count: " << running_flag.get_val_D2H();
    }
    sw.stop();

    printf("%s send count:%d iter:%d sssp done:%f\n", mode == 1 ? "Async" : "Sync", send_count.get_val_D2H(), iteration,
           sw.ms());
    if (FLAGS_output.size() > 0) {
        dev_graph_allocator.GatherDatum(node_distances);
        SSSPOutput(FLAGS_output.data(), node_distances.GetHostData());
//        dev_graph_allocator.GatherDatum(node_delta_distances);
//        SSSPOutput(FLAGS_output.data(), node_delta_distances.GetHostData());
    }
    return true;
}