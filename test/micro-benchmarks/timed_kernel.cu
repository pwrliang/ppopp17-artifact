#include "hip/hip_runtime.h"
// Groute: An Asynchronous Multi-GPU Programming Framework
// http://www.github.com/groute/groute
// Copyright (c) 2017, A. Barak
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice,
//   this list of conditions and the following disclaimer.
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
// * Neither the names of the copyright holders nor the names of its 
//   contributors may be used to endorse or promote products derived from this
//   software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
#include <chrono>
#include <cstdio>

#include <gtest/gtest.h>
#include <hip/hip_runtime.h>

#define REPETITIONS 100
#define MS_TIME 13

__global__ void Timed(unsigned long long clocks)
{
    unsigned long long target = clock64() + clocks;
    while(clock64() < target);
}

TEST(Microbenchmarks, TimedKernel)
{
    int dev = 0;
    hipDeviceProp_t props;
    hipGetDevice(&dev);
    hipGetDeviceProperties(&props, dev);

    float actual_time = ((float)MS_TIME * 1000.0f) *
        ((float)props.clockRate / 1024.0f);
    
    hipDeviceSynchronize();
    auto t1 = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < REPETITIONS; ++i)
    {
        Timed<<<500, 32>>>((unsigned long long)actual_time);
    }
    hipDeviceSynchronize();
    auto t2 = std::chrono::high_resolution_clock::now();

    double mstime = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count() / 1000.0 / REPETITIONS;

    printf("Kernel length: %f ms\n", mstime);
    
    ASSERT_LE(fabs(mstime - MS_TIME), 1.0f)
        << "The kernel took "
        << mstime << " ms instead of " << MS_TIME;
}

