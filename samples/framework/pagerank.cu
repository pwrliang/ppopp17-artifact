#include "hip/hip_runtime.h"
//
// Created by liang on 2/15/18.
//

#include "registry.h"
#include "kernel.h"

typedef float rank_t;

struct MyIterateKernel : public maiter::IterateKernel<rank_t, rank_t> {
    __forceinline__ __device__ rank_t InitValue(const index_t node, index_t out_degree) const {
//        printf("call %d\n", node);
        return 0;
    }

    __forceinline__ __device__ rank_t InitDelta(const index_t node, index_t out_degree) const {
        return 0.2;
    }

    __forceinline__ __device__ float accumulate(const rank_t a, const rank_t b) const {
        return a + b;
    }

    __forceinline__ __device__ float
    g_func(const float delta, const index_t weight,
           const index_t out_degree) const {
        return 0.8 * delta / out_degree;
    }

    __forceinline__ __device__ virtual float IdentityElement() const {
        return 0;
    }
};

__global__ void createFunc(maiter::IterateKernel<rank_t, rank_t> **baseFunc) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *baseFunc = new MyIterateKernel();
    }
}

bool PageRank() {
    maiter::MaiterKernel<rank_t, rank_t> *kernel = new maiter::MaiterKernel<rank_t, rank_t>();

    createFunc << < 1, 1 >> > (kernel->DeviceKernelObject());
    GROUTE_CUDA_CHECK(hipDeviceSynchronize());

    kernel->InitValue();

    kernel->DataDriven();

    delete kernel;
//    kernel->
    return true;
}