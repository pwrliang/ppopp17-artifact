#include "hip/hip_runtime.h"
//
// Created by liang on 2/16/18.
//
#include <vector>
#include <algorithm>
#include <thread>
#include <memory>
#include <random>
#include <hip/hip_runtime.h>
#include <>
#include <groute/event_pool.h>
#include <groute/graphs/csr_graph.h>
#include <groute/dwl/work_source.cuh>
#include <groute/device/cta_scheduler.cuh>
#include <utils/parser.h>
#include <utils/utils.h>
#include <utils/graphs/traversal.h>
#include <utils/stopwatch.h>
#include <moderngpu/context.hxx>
#include <moderngpu/kernel_scan.hxx>
#include <gflags/gflags.h>
#include <glog/logging.h>
#include <boost/format.hpp>
#include <utils/cuda_utils.h>
#include "pr_common.h"

DECLARE_double(wl_alloc_factor);
DECLARE_uint64(wl_alloc_abs);
DECLARE_int32(max_pr_iterations);
DECLARE_double(threshold);
DECLARE_int32(grid_size);
DECLARE_int32(block_size);
DECLARE_double(epsilon);
DECLARE_bool(cta_np);
DEFINE_int32(first_iteration, 20, "Iteration times for Topology-Driven");

namespace hybrid_unopt_pr {
    template<typename WorkSource,
            typename TGraph,
            template<typename> class RankDatum,
            template<typename> class ResidualDatum>
    __global__ void PageRankInit__Single__(
            WorkSource work_source,
            TGraph graph,
            RankDatum<rank_t> current_ranks, ResidualDatum<rank_t> residual) {
        unsigned tid = TID_1D;
        unsigned nthreads = TOTAL_THREADS_1D;

        for (index_t ii = 0 + tid; ii < work_source.get_size(); ii += nthreads) {
            index_t node = work_source.get_work(ii);

            current_ranks[node] = 1.0 - ALPHA;

            index_t
                    begin_edge = graph.begin_edge(node),
                    end_edge = graph.end_edge(node),
                    out_degree = end_edge - begin_edge;

            if (out_degree == 0) continue;

            rank_t update = ((1.0 - ALPHA) * ALPHA) / out_degree;

            for (index_t edge = begin_edge; edge < end_edge; ++edge) {
                index_t dest = graph.edge_dest(edge);
                atomicAdd(residual.get_item_ptr(dest), update);
            }
        }
    }

    template<
            typename WorkSource,
            typename TGraph, template<typename> class RankDatum,
            template<typename> class ResidualDatum>
    __global__ void PageRankKernelTopologyDriven__Single__(
            WorkSource work_source, TGraph graph,
            RankDatum<rank_t> current_ranks, ResidualDatum<rank_t> residual) {
        uint32_t tid = TID_1D;
        uint32_t nthreads = TOTAL_THREADS_1D;

        uint32_t work_size = work_source.get_size();

        for (uint32_t i = 0 + tid; i < work_size; i += nthreads) {
            index_t node = work_source.get_work(i);

            rank_t res = atomicExch(residual.get_item_ptr(node), 0);
            if (res == 0) continue; // might happen if work_source has duplicates

            current_ranks[node] += res;

            index_t
                    begin_edge = graph.begin_edge(node),
                    end_edge = graph.end_edge(node),
                    out_degree = end_edge - begin_edge;

            if (out_degree == 0) continue;

            rank_t update = res * ALPHA / out_degree;

            for (index_t edge = begin_edge; edge < end_edge; ++edge) {
                index_t dest = graph.edge_dest(edge);
                atomicAdd(residual.get_item_ptr(dest), update);
            }
        }
    }

    template<
            typename WorkSource,
            typename TGraph, template<typename> class RankDatum,
            template<typename> class ResidualDatum>
    __global__ void PageRankKernelTopologyDrivenCTA__Single__(
            WorkSource work_source, TGraph graph,
            RankDatum<rank_t> current_ranks, ResidualDatum<rank_t> residual) {
        uint32_t tid = TID_1D;
        uint32_t nthreads = TOTAL_THREADS_1D;
        uint32_t work_size = work_source.get_size();
        uint32_t work_size_rup = round_up(work_size, blockDim.x) * blockDim.x;

        for (uint32_t i = 0 + tid; i < work_size_rup; i += nthreads) {
            groute::dev::np_local<rank_t> local_work = {0, 0, 0.0};

            if (i < work_size) {
                index_t node = work_source.get_work(i);

                rank_t res = atomicExch(residual.get_item_ptr(node), 0);

                if (res > 0) {
                    current_ranks[node] += res;

                    local_work.start = graph.begin_edge(node);
                    local_work.size = graph.end_edge(node) - local_work.start;
                    if (local_work.size > 0) {
                        rank_t update = res * ALPHA / local_work.size;

                        local_work.meta_data = update;
                    }
                }
            }

            groute::dev::CTAWorkScheduler<rank_t>::template schedule(
                    local_work,
                    [&graph, &residual](index_t edge, rank_t update) {
                        index_t dest = graph.edge_dest(edge);
                        atomicAdd(residual.get_item_ptr(dest), update);
                    }
            );
        }
    }

    template<
            typename WorkSource, typename WorkTarget,
            typename TGraph, template<typename> class RankDatum,
            template<typename> class ResidualDatum>
    __global__ void PageRankKernelDataDriven__Single__(
            WorkSource work_source, WorkTarget work_target,
            float EPSILON, TGraph graph,
            RankDatum<rank_t> current_ranks, ResidualDatum<rank_t> residual) {
        uint32_t tid = TID_1D;
        uint32_t nthreads = TOTAL_THREADS_1D;

        uint32_t work_size = work_source.get_size();

        for (uint32_t i = 0 + tid; i < work_size; i += nthreads) {
            index_t node = work_source.get_work(i);

            rank_t res = atomicExch(residual.get_item_ptr(node), 0);
            if (res == 0) continue; // might happen if work_source has duplicates

            current_ranks[node] += res;

            index_t
                    begin_edge = graph.begin_edge(node),
                    end_edge = graph.end_edge(node),
                    out_degree = end_edge - begin_edge;

            if (out_degree == 0) continue;

            rank_t update = res * ALPHA / out_degree;

            for (index_t edge = begin_edge; edge < end_edge; ++edge) {
                index_t dest = graph.edge_dest(edge);
                rank_t prev = atomicAdd(residual.get_item_ptr(dest), update);

                if (prev <= EPSILON && prev + update > EPSILON) {
                    work_target.append(dest);
                }
            }
        }
    }

    template<
            typename WorkSource, typename WorkTarget,
            typename TGraph, template<typename> class RankDatum,
            template<typename> class ResidualDatum>
    __global__ void PageRankKernelDataDrivenCTA__Single__(
            WorkSource work_source, WorkTarget work_target,
            float EPSILON, TGraph graph,
            RankDatum<rank_t> current_ranks, ResidualDatum<rank_t> residual) {
        uint32_t tid = TID_1D;
        uint32_t nthreads = TOTAL_THREADS_1D;

        uint32_t work_size = work_source.get_size();
        uint32_t work_size_rup = round_up(work_size, blockDim.x) * blockDim.x;

        for (uint32_t i = 0 + tid; i < work_size_rup; i += nthreads) {

            groute::dev::np_local<rank_t> local_work = {0, 0, 0.0};

            if (i < work_size) {
                index_t node = work_source.get_work(i);
                rank_t res = atomicExch(residual.get_item_ptr(node), 0);

                if (res > 0) {
                    current_ranks[node] += res;
                    local_work.start = graph.begin_edge(node);
                    local_work.size = graph.end_edge(node) - local_work.start;

                    index_t
                            begin_edge = graph.begin_edge(node),
                            end_edge = graph.end_edge(node),
                            out_degree = end_edge - begin_edge;
                    if (local_work.size > 0) {
                        rank_t update = res * ALPHA / out_degree;

                        local_work.meta_data = update;
                    }
                }
            }

            groute::dev::CTAWorkScheduler<rank_t>::template schedule(
                    local_work,
                    [&work_target, &graph, &residual, &EPSILON](index_t edge, rank_t update) {
                        index_t dest = graph.edge_dest(edge);
                        rank_t prev = atomicAdd(residual.get_item_ptr(dest), update);

                        if (prev <= EPSILON && prev + update > EPSILON) {
                            work_target.append(dest);
                        }
                    }
            );
        }
    }

    /*
    * The per-device Page Rank problem
    */
    template<typename TGraph,
            template<typename> class ResidualDatum,
            template<typename> class RankDatum>
    struct Problem {
        TGraph m_graph;
        ResidualDatum<rank_t> m_residual;
        RankDatum<rank_t> m_current_ranks;

        Problem(const TGraph &graph, RankDatum<rank_t> current_ranks, ResidualDatum<rank_t> residual) :
                m_graph(graph), m_residual(residual), m_current_ranks(current_ranks) {
        }

        template<typename WorkSource>
        void Init__Single__(const WorkSource &workSource, groute::Stream &stream) const {
            dim3 grid_dims, block_dims;
            KernelSizing(grid_dims, block_dims, m_graph.owned_nnodes());

            Marker::MarkWorkitems(m_graph.owned_nnodes(), "PageRankInit__Single__");

            PageRankInit__Single__ << < grid_dims, block_dims, 0, stream.hip_stream >> >
                                                                  (workSource, m_graph, m_current_ranks, m_residual);
        }

        template<typename WorkSource>
        void
        RelaxTopologyDriven__Single__(const WorkSource &work_source, groute::Stream &stream) {
            dim3 grid_dims, block_dims;
            KernelSizing(grid_dims, block_dims, work_source.get_size());

            Marker::MarkWorkitems(work_source.get_size(), "PageRankKernelTopologyDriven__Single__");

            if (FLAGS_cta_np)
                PageRankKernelTopologyDrivenCTA__Single__ << < grid_dims, block_dims, 0, stream.hip_stream >> >
                                                                                         (work_source, m_graph, m_current_ranks, m_residual);
            else
                PageRankKernelTopologyDriven__Single__ << < grid_dims, block_dims, 0, stream.hip_stream >> >
                                                                                      (work_source, m_graph, m_current_ranks, m_residual);
        }

        template<typename WorkSource,
                typename WorkTarget>
        void
        RelaxDataDriven__Single__(const WorkSource &work_source, WorkTarget &output_worklist, groute::Stream &stream) {
            dim3 grid_dims, block_dims;
            KernelSizing(grid_dims, block_dims, work_source.get_size());

            float EPSILON = FLAGS_epsilon;
            Marker::MarkWorkitems(work_source.get_size(), "PageRankKernel__Single__");

            if (FLAGS_cta_np)
                PageRankKernelDataDrivenCTA__Single__ << < grid_dims, block_dims, 0, stream.hip_stream >> >
                                                                                     (work_source, output_worklist.DeviceObject(), EPSILON, m_graph, m_current_ranks, m_residual);
            else
                PageRankKernelDataDriven__Single__ << < grid_dims, block_dims, 0, stream.hip_stream >> >
                                                                                  (work_source, output_worklist.DeviceObject(), EPSILON, m_graph, m_current_ranks, m_residual);
        }
    };

    struct Algo {
        static const char *NameLower() { return "pr"; }

        static const char *Name() { return "PR"; }


        template<
                typename TGraphAllocator, typename ResidualDatum, typename RankDatum, typename...UnusedData>
        static const std::vector<rank_t> &Gather(
                TGraphAllocator &graph_allocator, ResidualDatum &residual, RankDatum &current_ranks,
                UnusedData &... data) {
            graph_allocator.GatherDatum(current_ranks);
            return current_ranks.GetHostData();
        }

        template<
                typename ResidualDatum, typename RankDatum, typename...UnusedData>
        static std::vector<rank_t> Host(
                groute::graphs::host::CSRGraph &graph, ResidualDatum &residual, RankDatum &current_ranks,
                UnusedData &... data) {
            return PageRankHost(graph);
        }

        static int Output(const char *file, const std::vector<rank_t> &ranks) {
            return PageRankOutput(file, ranks);
        }

        static int CheckErrors(std::vector<rank_t> &ranks, std::vector<rank_t> &regression) {
            return PageRankCheckErrors(ranks, regression);
        }
    };
}

bool HybridDrivenPR() {
    VLOG(0) << "HybridDrivenPR";
    if (FLAGS_cta_np)
        VLOG(0) << "CTA_NP Enabled";
    typedef groute::Queue<index_t> Worklist;
    groute::graphs::single::NodeOutputDatum<rank_t> residual;
    groute::graphs::single::NodeOutputDatum<rank_t> current_ranks;

    utils::traversal::Context<hybrid_unopt_pr::Algo> context(1);

    groute::graphs::single::CSRGraphAllocator
            dev_graph_allocator(context.host_graph);

    context.SetDevice(0);

    dev_graph_allocator.AllocateDatumObjects(residual, current_ranks);

    context.SyncDevice(0); // graph allocations are on default streams, must sync device

    groute::Stream stream = context.CreateStream(0);

    mgpu::standard_context_t mgpu_context(true, stream.hip_stream);

    hybrid_unopt_pr::Problem<
            groute::graphs::dev::CSRGraph,
            groute::graphs::dev::GraphDatum, groute::graphs::dev::GraphDatum>
            solver(
            dev_graph_allocator.DeviceObject(),
            current_ranks.DeviceObject(),
            residual.DeviceObject());

    size_t max_work_size = context.host_graph.nedges * FLAGS_wl_alloc_factor;

    if (FLAGS_wl_alloc_abs > 0)
        max_work_size = FLAGS_wl_alloc_abs;

    Worklist wl1(max_work_size, 0, "input queue"), wl2(max_work_size, 0, "output queue");

    wl1.ResetAsync(stream.hip_stream);
    wl2.ResetAsync(stream.hip_stream);
    stream.Sync();

    Worklist *in_wl = &wl1, *out_wl = &wl2;

    Stopwatch sw(true);

    solver.Init__Single__(groute::dev::WorkSourceRange<index_t>(
            dev_graph_allocator.DeviceObject().owned_start_node(),
            dev_graph_allocator.DeviceObject().owned_nnodes()), stream);

    int iteration = 0;

    for (iteration = 0; iteration < FLAGS_first_iteration; iteration++) {
        solver.RelaxTopologyDriven__Single__(
                groute::dev::WorkSourceRange<index_t>(dev_graph_allocator.DeviceObject().owned_start_node(),
                                                      dev_graph_allocator.DeviceObject().owned_nnodes()), stream);
        stream.Sync();
        VLOG(1) << "Topology-Driven Iteration: " << iteration;
    }

    solver.RelaxDataDriven__Single__(
            groute::dev::WorkSourceRange<index_t>(dev_graph_allocator.DeviceObject().owned_start_node(),
                                                  dev_graph_allocator.DeviceObject().owned_nnodes()),
            *in_wl, stream);

    groute::Segment<index_t> work_seg;

    work_seg = in_wl->GetSeg(stream);

    while (work_seg.GetSegmentSize() > 0) {
        solver.RelaxDataDriven__Single__(groute::dev::WorkSourceArray<index_t>(work_seg.GetSegmentPtr(),
                                                                               work_seg.GetSegmentSize()),
                                         *out_wl, stream);
        VLOG(1) << "Data-Driven Iteration: " << iteration << " INPUT " << work_seg.GetSegmentSize() << " OUTPUT "
                << out_wl->GetCount(stream);

        if (++iteration > FLAGS_max_pr_iterations) {
            LOG(WARNING) << "maximum iterations reached";
            break;
        }

        in_wl->ResetAsync(stream.hip_stream);
        std::swap(in_wl, out_wl);
        work_seg = in_wl->GetSeg(stream);
    }

    sw.stop();

    VLOG(1)
    << boost::format("%s terminated after %d iterations (max: %d)") % hybrid_unopt_pr::Algo::Name() % iteration %
       FLAGS_max_pr_iterations;
    VLOG(0) << hybrid_unopt_pr::Algo::Name() << ": " << sw.ms() << " ms. <filter>";
    // Gather
    auto gathered_output = hybrid_unopt_pr::Algo::Gather(dev_graph_allocator, residual, current_ranks);

    if (FLAGS_output.length() != 0)
        hybrid_unopt_pr::Algo::Output(FLAGS_output.c_str(), gathered_output);

    if (FLAGS_check) {
        auto regression = hybrid_unopt_pr::Algo::Host(context.host_graph, residual, current_ranks);
        return hybrid_unopt_pr::Algo::CheckErrors(gathered_output, regression) == 0;
    } else {
        LOG(WARNING) << "Result not checked";
        return true;
    }
}